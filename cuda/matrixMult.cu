#include <iostream>
#include <hip/hip_runtime.h>

#define N 16 // Matrix size (N x N)
/*
blockIdx.x - block index in the x-dimension, there is also blockIdx.y and blockIdx.z

*/
__global__ void matrixMul(int *A, int *B, int *C, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < n && col < n) { 
        int sum = 0;
        for (int k = 0; k < n; k++) {
            sum += A[row * n + k] * B[k * n + col];
        }
        C[row * n + col] = sum;
    }
}

void printMatrix(int *mat, int n) {
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            std::cout << mat[i * n + j] << " ";
        }
        std::cout << std::endl;
    }
}

int main() {
    int size = N * N * sizeof(int);
    int *h_A, *h_B, *h_C, *d_A, *d_B, *d_C;

    // Allocate memory on host
    h_A = (int*)malloc(size);
    h_B = (int*)malloc(size);
    h_C = (int*)malloc(size);

    // Initialize matrices A and B
    for (int i = 0; i < N * N; i++) {
        h_A[i] = rand() % 10;
        h_B[i] = rand() % 10;
    }

    // Allocate memory on GPU
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    // Copy data from host to device
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Define block and grid sizes
    /*
    dim3 - Cuda struct that defines block dimensions
    */
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((N + threadsPerBlock.x - 1) / threadsPerBlock.x, 
                   (N + threadsPerBlock.y - 1) / threadsPerBlock.y);

    // Launch kernel
    matrixMul<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C, N);
    hipDeviceSynchronize();

    // Copy result back to host
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Print matrices
    std::cout << "Matrix A:\n";
    printMatrix(h_A, N);
    std::cout << "Matrix B:\n";
    printMatrix(h_B, N);
    std::cout << "Matrix C (Result):\n";
    printMatrix(h_C, N);

    // Free memory
    free(h_A); free(h_B); free(h_C);
    hipFree(d_A); hipFree(d_B); hipFree(d_C);

    return 0;
}
